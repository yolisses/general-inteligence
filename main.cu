
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIZE 10

#define cudaCheckError()                                                                       \
    {                                                                                          \
        hipError_t e = hipGetLastError();                                                    \
        if (e != hipSuccess)                                                                  \
        {                                                                                      \
            printf("\nCuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        }                                                                                      \
    }

struct Node;
__device__ Node *nodes;

struct Node
{
    Node *childs[4];

    __device__ int id()
    {
        return (int)(this - nodes);
    }
};

__global__ void allocNodes()
{
    hipMalloc(&nodes, sizeof(Node) * SIZE);
}

__global__ void initializeRandom()
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    hiprandState_t state;
    hiprand_init(clock64(), i * blockDim.x + j, 0, &state);

    nodes[i].childs[j] = &nodes[hiprand(&state) % SIZE];

    // printf("nodes[%d].child[%d] = %d\n", i, j, nodes[i].childs[j]->id());
}

int main()
{
    allocNodes<<<1, 1>>>();
    hipDeviceSynchronize();
    initializeRandom<<<SIZE, 4>>>();
    hipDeviceSynchronize();

    cudaCheckError();
}

//UriSE tenha DETERMINAÇÃO